// Get from: https://developer.nvidia.com/blog/using-shared-memory-cuda-cc/

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void dynamicReverse(int *d, int n)
{
  extern __shared__ int s[];
  int t = threadIdx.x;
  int tr = n-t-1;
  s[t] = d[t];
  __syncthreads();
  d[t] = s[tr];
}

int main()
{
  const int n = 64;
  int a[n], r[n], d[n];

  for (int i = 0; i < n; i++) {
    a[i] = i;
    r[i] = n-i-1;
    d[i] = 0;
  }

  int *d_d;
  hipMalloc(&d_d, n * sizeof(int));

  // run version with static shared memory
  hipMemcpy(d_d, a, n*sizeof(int), hipMemcpyHostToDevice);
  dynamicReverse<<<1,n,n*sizeof(int)>>>(d_d, n);
  hipMemcpy(d, d_d, n*sizeof(int), hipMemcpyDeviceToHost);
  for (int i = 0; i < n; i++)
    if (d[i] != r[i]) {
      printf("Error: d[%d]!=r[%d] (%d, %d)n", i, i, d[i], r[i]);
      exit(1);
    }
    printf("PASS\n");
    return 0;
}
