
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <hipblas.h>

int main() {
    int numRows = 1024;
    int numCols = 1024;
    size_t bytes = numRows * numCols * sizeof(float);
    float alpha = 1.0f;
    float beta = 1.0f;

    printf("Allocating memory for matrices on host...\n");

    // Allocate memory for matrices on host
    float *h_A = (float*)malloc(bytes);
    float *h_B = (float*)malloc(bytes);
    float *h_C = (float*)malloc(bytes);

    printf("Initializing matrices on host...\n");

    // Initialize matrices on host
    for(int i = 0; i < numRows * numCols; i++) {
        h_A[i] = 1.0f; // Example values
        h_B[i] = 2.0f;
    }

    printf("Allocating memory for matrices on device...\n");

    // Allocate memory for matrices on device
    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, bytes);
    hipMalloc(&d_B, bytes);
    hipMalloc(&d_C, bytes);

    printf("Copying matrices from host to device...\n");

    // Copy matrices from host to device
    hipMemcpy(d_A, h_A, bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, bytes, hipMemcpyHostToDevice);

    // Create cuBLAS context
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    printf("Performing matrix addition...\n");

    // Perform matrix addition: C = alpha*A + beta*B
    hipblasSgeam(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, numRows, numCols, &alpha, d_A, numRows, &beta, d_B, numRows, d_C, numRows);

    printf("Copying result to host...\n");

    // Copy result back to host
    hipMemcpy(h_C, d_C, bytes, hipMemcpyDeviceToHost);

    printf("Cleaning up...\n");

    // Cleanup
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    free(h_A);
    free(h_B);
    free(h_C);

    hipblasDestroy(handle);

    printf("Done.\n");

    return 0;
}